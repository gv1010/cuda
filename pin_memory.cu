#include <iostream>
#include <hip/hip_runtime.h>


int main(){
    int N = 1048576;
    int bytes = N * sizeof(int);
    int *h_a = (int *)malloc(bytes);
    int *h_aPinned;
    int *d_a;

    hipHostMalloc((void**)&h_aPinned, bytes, hipHostMallocDefault) ;
    hipMalloc((void**)&d_a, bytes);

    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf(" Pageable Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

    hipEvent_t pstartEvent, pstopEvent; 
    hipEventCreate(&pstartEvent);
    hipEventCreate(&pstopEvent);
    hipEventRecord(pstartEvent, 0);
    hipMemcpy(d_a, h_aPinned, bytes, hipMemcpyHostToDevice);
    hipEventRecord(pstopEvent, 0);
    hipEventSynchronize(pstopEvent);

    float ptime;
    hipEventElapsedTime(&ptime, pstartEvent, pstopEvent);
    printf(" Pinned Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / ptime);
    
    hipHostFree(h_aPinned);
    hipFree(d_a);
    free(h_a);
    return 0;
}
