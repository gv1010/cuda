#include "hip/hip_runtime.h"
%%writefile shared_mat_mul.cu

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void navie_mat_mul(float *P_A, float *P_B,float *P_C, int Width){
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < Width && col < Width){
        for (int k=0; k<Width; k++){
            P_C[row*Width+col] += P_A[row*Width+k] * P_B[k*Width+col];
        }
    }
}

__global__ void  mat_mul(float *A, float *B, float *C, int width){
    __shared__ float As[32][32];
    __shared__ float Bs[32][32];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx + blockDim.x + tx;

    float element_value = 0;

    As[ty][tx] = A[row*width + tx];
    Bs[ty][tx] = B[ty*width + col];

    __syncthreads();

    for (int k=0; k < blockDim.x; k++){
        element_value += As[ty][k]* Bs[k][tx];
    }
    
    C[row*width + col] = element_value;


}


int main(){
    int rows = 2048;
    int cols = 2048;
    int size = rows * cols; 
    float *hA, *hB, *hC; // Host arrays
    float *dA, *dB, *dC; // Device arrays

    // Allocate memory on host
    hA = new float[size];
    hB = new float[size];
    hC = new float[size];

    // Allocate memory on device
    hipMalloc((void **)&dA, size * sizeof(float));
    hipMalloc((void **)&dB, size * sizeof(float));
    hipMalloc((void **)&dC, size * sizeof(float));



    // Initialize host arrays (example values)
    for (int row = 0; row < rows; row ++){
        for (int col = 0; col < cols; col++){
            hA[row*cols + col] = row*cols + col;
        }
    }

    // Initialize host arrays (example values)
    for (int row = 0; row < rows; row ++){
        for (int col = 0; col < cols; col++){
            hB[row*cols + col] = row*cols + col;
        }
    }

    // Copy data from host to device
    hipMemcpy(dA, hA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size * sizeof(float), hipMemcpyHostToDevice);


    // Kernel launch configuration
    dim3 blockSize(32, 32, 1);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y, 1);

    printf("%25s%25s%25s\n", "Exectution", "Bandwidth (GB/s)", "GPU Time (ms)");
    printf("\n");

    printf("%25s","Matrix Mul");

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    float total_milliseconds1 = 0;
    int num_runs = 100;

    for (int i=0; i < num_runs; i++){
        hipEventRecord(start1);
        mat_mul<<<gridSize, blockSize>>>(dA, dB, dC, rows);
        hipEventRecord(stop1);
        hipEventSynchronize(stop1);
        float milliseconds1 = 0;
        hipEventElapsedTime(&milliseconds1, start1, stop1);
        total_milliseconds1 += milliseconds1;
    }
    float average_milliseconds1 = total_milliseconds1 / num_runs;
    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    hipMemcpy(hC, dC, size * sizeof(float), hipMemcpyDeviceToHost);
    printf("%20.4f%25.4f\n", 2 * size * sizeof(float)* 1e-6 / average_milliseconds1, average_milliseconds1 );



    printf("%25s","Navie Matrix Mul");

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    float total_milliseconds2 = 0;
    int num_runs1 = 100;

    for (int i=0; i < num_runs1; i++){
        hipEventRecord(start2);
        navie_mat_mul<<<gridSize, blockSize>>>(dA, dB, dC, rows);
        hipEventRecord(stop2);
        hipEventSynchronize(stop2);
        float milliseconds2 = 0;
        hipEventElapsedTime(&milliseconds2, start2, stop2);
        total_milliseconds2 += milliseconds2;
    }
    float average_milliseconds2 = total_milliseconds2 / num_runs1;
    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    hipMemcpy(hC, dC, size * sizeof(float), hipMemcpyDeviceToHost);
    printf("%20.4f%25.4f\n", 2 * size * sizeof(float)* 1e-6 / average_milliseconds2, average_milliseconds2 );
    return 0;

}


//  Exectution         Bandwidth (GB/s)            GPU Time (ms)

// Matrix Mul             46.1203                   0.7275
// Navie Matrix Mul        0.4569                  73.4422

